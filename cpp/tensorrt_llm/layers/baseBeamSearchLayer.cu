#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "tensorrt_llm/common/cudaUtils.h"
#include "tensorrt_llm/common/memoryUtils.h"
#include "tensorrt_llm/kernels/beamSearchPenaltyKernels.h"
#include "tensorrt_llm/layers/baseBeamSearchLayer.h"
#include "tensorrt_llm/layers/fillBuffers.h"

#include <algorithm>

using namespace tensorrt_llm::common;
using namespace tensorrt_llm::kernels;

namespace tensorrt_llm
{
namespace layers
{

__global__ void update_indir_cache_kernel(int* tgt_indir_cache, const int* src_indir_cache, const int** parent_ids,
    const bool* finished, const int* sequence_lengths, const int* input_lengths, int batch_dim, int local_batch_size,
    int beam_width, int max_kv_cache_length, int max_seq_len)
{
    int time_step = threadIdx.x + blockIdx.x * blockDim.x;
    int bb_id = threadIdx.y + blockIdx.y * blockDim.y;   // should be just blockIdx.y?
    const int current_step{sequence_lengths[bb_id] - 1}; // the sequence_lengths is updated, need to minus 1
    const int batch_id = bb_id / beam_width;
    const int beam_id = bb_id % beam_width;
    if (bb_id >= beam_width * local_batch_size || time_step < (max_seq_len - max_kv_cache_length) || finished[bb_id])
    {
        return;
    }
    int time_step_circ = time_step % max_kv_cache_length;

    // for the parent_ids, we will still keep it for all past tokens (i.e. max_seq_len)
    const int src_beam = parent_ids[batch_id][beam_id * max_seq_len + current_step];

    // for the indir tables, we have the cyclic kv cache.
    const uint32_t tgt_offset
        = batch_id * beam_width * max_kv_cache_length + beam_id * max_kv_cache_length + time_step_circ;
    const uint32_t src_offset
        = batch_id * beam_width * max_kv_cache_length + src_beam * max_kv_cache_length + time_step_circ;

    tgt_indir_cache[tgt_offset] = (time_step == current_step) ? beam_id : src_indir_cache[src_offset];
}

void update_indir_cache_kernelLauncher(int* tgt_indir_cache, const int* src_indir_cache, const int** parent_ids,
    const bool* finished, const int* sequence_lengths, const int* input_lengths, int batch_dim, int local_batch_size,
    int beam_width, int max_seq_len, int max_kv_cache_length, hipStream_t stream)
{
    const dim3 block(32);
    // Update indirections steps [input_length[bb_id], sequence_lengths[bb_id]], included
    const dim3 grid((max_seq_len + block.x - 1) / block.x, local_batch_size * beam_width);
    update_indir_cache_kernel<<<grid, block, 0, stream>>>(tgt_indir_cache, src_indir_cache, parent_ids, finished,
        sequence_lengths, input_lengths, batch_dim, local_batch_size, beam_width, max_kv_cache_length, max_seq_len);
}

template <typename T>
BaseBeamSearchLayer<T>::BaseBeamSearchLayer(size_t vocab_size, size_t vocab_size_padded, hipStream_t stream,
    IAllocator* allocator, bool is_free_buffer_after_forward)
    : BaseLayer(stream, allocator, is_free_buffer_after_forward, nullptr)
    , vocab_size_(vocab_size)
    , vocab_size_padded_(vocab_size_padded)
{
}

template <typename T>
BaseBeamSearchLayer<T>::BaseBeamSearchLayer(BaseBeamSearchLayer<T> const& beam_search_layer)
    : BaseLayer(beam_search_layer)
    , vocab_size_(beam_search_layer.vocab_size_)
    , vocab_size_padded_(beam_search_layer.vocab_size_padded_)
    , topk_softmax_workspace_size_(beam_search_layer.topk_softmax_workspace_size_)
{
}

template <typename T>
BaseBeamSearchLayer<T>::~BaseBeamSearchLayer()
{
    TLLM_LOG_DEBUG(__PRETTY_FUNCTION__);
    freeBuffer();
}

template <typename T>
void BaseBeamSearchLayer<T>::freeBuffer()
{
    TLLM_LOG_DEBUG("%s start", __PRETTY_FUNCTION__);
    if (is_allocate_buffer_)
    {
        allocator_->free((void**) (&temperature_buf_));
        allocator_->free((void**) (&min_lengths_buf_));
        allocator_->free((void**) (&repetition_penalty_buf_));
        is_allocate_buffer_ = false;
    }
    TLLM_LOG_DEBUG("%s stop", __PRETTY_FUNCTION__);
}

template <typename T>
void BaseBeamSearchLayer<T>::allocateBuffer(size_t batch_size)
{
    TLLM_LOG_DEBUG("%s start", __PRETTY_FUNCTION__);
    temperature_buf_ = allocator_->reMalloc(temperature_buf_, sizeof(float) * batch_size, false);
    min_lengths_buf_ = allocator_->reMalloc(min_lengths_buf_, sizeof(int) * batch_size, false);
    repetition_penalty_buf_ = allocator_->reMalloc(repetition_penalty_buf_, sizeof(float) * batch_size, false);

    is_allocate_buffer_ = true;
    TLLM_LOG_DEBUG("%s stop", __PRETTY_FUNCTION__);
}

template <typename T>
void BaseBeamSearchLayer<T>::setupBase(size_t batch_size, SetupParams const& setupParams)
{
    allocateBuffer(batch_size);
    TLLM_LOG_DEBUG("%s start", __PRETTY_FUNCTION__);
    // Setup penalties.
    FillBuffers const fillBuffers{batch_size, stream_};

    fillBuffers(setupParams.temperature, 1.0f, mTemperature, temperature_buf_);
    fillBuffers(setupParams.min_length, 1, mMinLength, min_lengths_buf_);

    mRepetitionPenaltyType = RepetitionPenaltyType::None;
    if (setupParams.repetition_penalty || setupParams.presence_penalty)
    {
        TLLM_CHECK_WITH_INFO(!(setupParams.repetition_penalty && setupParams.presence_penalty),
            "Found ambiguous parameters repetition_penalty and presence_penalty "
            "which are mutually exclusive. "
            "Please provide one of repetition_penalty or presence_penalty.");
        mRepetitionPenaltyType
            = setupParams.repetition_penalty ? RepetitionPenaltyType::Multiplicative : RepetitionPenaltyType::Additive;
        if (mRepetitionPenaltyType == RepetitionPenaltyType::Multiplicative)
        {
            fillBuffers(setupParams.repetition_penalty, 1.0f, mRepetitionPenalty, repetition_penalty_buf_);
        }
        else
        {
            fillBuffers(setupParams.presence_penalty, 1.0f, mRepetitionPenalty, repetition_penalty_buf_);
        }
    }
    TLLM_LOG_DEBUG("%s stop", __PRETTY_FUNCTION__);
}

template <typename T>
void BaseBeamSearchLayer<T>::forward(BeamSearchOutputParams& outputs, ForwardParams const& params)
{
    TLLM_LOG_DEBUG("%s", __PRETTY_FUNCTION__);
    Tensor& output_ids_ptr = outputs.output_ids_ptr;

    const auto batch_size = static_cast<std::int32_t>(output_ids_ptr.shape[0]);
    const auto beam_width = static_cast<std::int32_t>(output_ids_ptr.shape[1]);
    const auto max_seq_len = static_cast<std::int32_t>(output_ids_ptr.shape[2]);

    TLLM_CHECK_WITH_INFO(params.ite == 0, "Pipeline Parallelism is not supported yet !");

    const int ite{params.ite};
    Tensor const& logits = params.logits;
    const auto local_batch_size = logits.shape[0];

    const T* embedding_bias = params.embedding_bias ? params.embedding_bias->template getPtr<const T>() : nullptr;

    auto* end_ids = params.end_ids.template getPtr<const int>();
    auto* const input_lengths = params.input_lengths ? params.input_lengths->template getPtr<const int>() : nullptr;
    int* sequence_length = (outputs.sequence_length) ? outputs.sequence_length->template getPtr<int>() : nullptr;

    invokeAddBiasApplyPenalties(logits.getPtr<T>(), output_ids_ptr.template getPtr<const int*>(),
        outputs.parent_ids_ptr.template getPtr<const int*>(), input_lengths, sequence_length, embedding_bias, ite,
        local_batch_size, batch_size, beam_width, vocab_size_, vocab_size_padded_, end_ids, temperature_buf_,
        mTemperature, repetition_penalty_buf_, mRepetitionPenalty, mRepetitionPenaltyType, min_lengths_buf_,
        max_seq_len, stream_);
    sync_check_cuda_error();

    invokeSoftMax(outputs, params);

    if (beam_width > 1)
    {
        update_indir_cache_kernelLauncher(outputs.tgt_cache_indirection.template getPtr<int>(),
            params.src_cache_indirection.template getPtr<const int>(),
            outputs.parent_ids_ptr.template getPtr<const int*>(), outputs.finished->template getPtr<const bool>(),
            sequence_length, input_lengths, batch_size, local_batch_size, beam_width, max_seq_len,
            params.max_kv_cache_length, stream_);
        sync_check_cuda_error();
    }
    sync_check_cuda_error();
    if (is_free_buffer_after_forward_)
    {
        freeBuffer();
    }
    sync_check_cuda_error();
}

template class BaseBeamSearchLayer<float>;
template class BaseBeamSearchLayer<half>;

} // namespace layers
} // namespace tensorrt_llm
