#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <assert.h>
#include <float.h>

#include "tensorrt_llm/common/assert.h"
#include "tensorrt_llm/common/cudaUtils.h"
#include "tensorrt_llm/kernels/samplingPenaltyKernels.h"

namespace tensorrt_llm
{
namespace kernels
{

// TODO Add half2 implementation
template <typename T>
__global__ void applyTemperaturePenalty(T* logits, const T* bias, const float temperatureInverse, const int m,
    const int vocabSize, const int vocabSizePadded)
{
    const bool IS_FP16 = std::is_same<T, half>::value;
    const T MAX_T_VAL = (IS_FP16) ? 65504.F : FLT_MAX;
    for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < m * vocabSizePadded;
         index += blockDim.x * gridDim.x)
    {
        T biasVal = bias == nullptr ? (T) (0.0f) : bias[index % vocabSizePadded];
        if (index % vocabSizePadded < vocabSize)
        {
            logits[index] = (logits[index] + biasVal) * (T) temperatureInverse;
        }
        else
        {
            logits[index] = -MAX_T_VAL;
        }
    }
}

template <>
__global__ void applyTemperaturePenalty(half2* logits, const half2* bias, const float temperatureInverse,
    const int batchSize, const int vocabSize, const int vocabSizePaddeded)
{
    assert(vocabSize % 2 == 0);
    assert(vocabSizePaddeded % 2 == 0);
    const half2 maskVal = __float2half2_rn(-65504.0f);
    const half2 tempInv = __float2half2_rn(temperatureInverse);

    const int halfVocabSize = vocabSize / 2;
    const int halfVocabSizePaddeded = vocabSizePaddeded / 2;
    for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < batchSize * halfVocabSizePaddeded;
         index += blockDim.x * gridDim.x)
    {
        int vocabIdx = index % halfVocabSizePaddeded;
        half2 logit = vocabIdx < halfVocabSize ? __ldg(&logits[index]) : maskVal;
        if (vocabIdx < halfVocabSize)
        {
            if (bias != nullptr)
            {
                logit = __hadd2(logit, bias[vocabIdx]);
            }
            logits[index] = __hmul2(logit, tempInv);
        }
    }
}

template <typename T>
void invokeApplyTemperaturePenalty(T* logits, const T* bias, const float temperature, const int batchSize,
    const int vocabSize, const int vocabSizePadded, hipStream_t stream)
{
    dim3 block(min(vocabSizePadded, 1024));
    dim3 grid(min(batchSize * vocabSizePadded / block.x, 65536));
    const T temperatureInverse = (T) (1.f / (temperature + 1e-6f));
    if (std::is_same<T, half>::value && vocabSize % 2 == 0 && vocabSizePadded % 2 == 0)
    {
        applyTemperaturePenalty<<<grid, block, 0, stream>>>(reinterpret_cast<half2*>(logits),
            reinterpret_cast<const half2*>(bias), temperatureInverse, batchSize, vocabSize, vocabSizePadded);
    }
    else
    {
        applyTemperaturePenalty<T>
            <<<grid, block, 0, stream>>>(logits, bias, temperatureInverse, batchSize, vocabSize, vocabSizePadded);
    }
}

template void invokeApplyTemperaturePenalty(float* logits, const float* bias, const float temperature,
    const int batchSize, const int vocabSize, const int vocabSizePadded, hipStream_t stream);

template void invokeApplyTemperaturePenalty(half* logits, const half* bias, const float temperature,
    const int batchSize, const int vocabSize, const int vocabSizePadded, hipStream_t stream);

template <typename T>
__global__ void batchApplyTemperaturePenalty(T* logits, const T* bias, const float* temperatures, const int batchSize,
    const int vocabSize, const int vocabSizePadded)
{
    // TODO: Add macro or device function to get MAX_T_VAL.
    const bool IS_FP16 = std::is_same<T, half>::value;
    const T MAX_T_VAL = (IS_FP16) ? 65504.F : FLT_MAX;
    extern __shared__ float invTemperatures[];
    if (threadIdx.x < batchSize)
    {
        invTemperatures[threadIdx.x] = 1.0f / (temperatures[threadIdx.x] + 1e-6f);
    }
    __syncthreads();

    for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < batchSize * vocabSizePadded;
         index += blockDim.x * gridDim.x)
    {
        int batchIdx = index / vocabSizePadded;
        int vocabIdx = index % vocabSizePadded;
        T logit = (vocabIdx < vocabSize) ? logits[index] : -MAX_T_VAL;
        if (vocabIdx < vocabSize)
        {
            if (bias != nullptr)
            {
                logit += bias[vocabIdx];
            }
            logit *= invTemperatures[batchIdx];
        }
        logits[index] = logit;
    }
}

__global__ void batchApplyTemperaturePenalty_h2(half2* logits, const half2* bias, const float* temperatures,
    const int batchSize, const int vocabSize, const int vocabSizePaddeded)
{
    assert(vocabSize % 2 == 0);
    assert(vocabSizePaddeded % 2 == 0);
    extern __shared__ half2 h2InvTemperatures[];
    if (threadIdx.x < batchSize)
    {
        h2InvTemperatures[threadIdx.x] = __float2half2_rn(1.f / (temperatures[threadIdx.x] + 1e-6f));
    }
    __syncthreads();

    const half2 maskVal = __float2half2_rn(-65504.0f);
    const int halfVocabSize = vocabSize / 2;
    const int halfVocabSizePaddeded = vocabSizePaddeded / 2;
    for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < batchSize * halfVocabSizePaddeded;
         index += blockDim.x * gridDim.x)
    {
        int batchIdx = index / halfVocabSizePaddeded;
        int vocabIdx = index % halfVocabSizePaddeded;
        half2 logit = vocabIdx < halfVocabSize ? __ldg(&logits[index]) : maskVal;
        if (vocabIdx < halfVocabSize)
        {
            if (bias != nullptr)
            {
                logit = __hadd2(logit, bias[vocabIdx]);
            }
            logits[index] = __hmul2(logit, h2InvTemperatures[batchIdx]);
        }
    }
}

template <typename T>
void invokeBatchApplyTemperaturePenalty(T* logits, const T* bias, const float* temperatures, const int batchSize,
    const int vocabSize, const int vocabSizePadded, hipStream_t stream)
{
    TLLM_LOG_DEBUG("%s start", __PRETTY_FUNCTION__);
    dim3 block(min(vocabSizePadded, 1024));
    dim3 grid(min(batchSize * vocabSizePadded / block.x, 65536));
    if (std::is_same<T, half>::value && vocabSize % 2 == 0 && vocabSizePadded % 2 == 0)
    {
        size_t smemSize = sizeof(half2) * batchSize;
        batchApplyTemperaturePenalty_h2<<<grid, block, smemSize, stream>>>(reinterpret_cast<half2*>(logits),
            reinterpret_cast<const half2*>(bias), temperatures, batchSize, vocabSize, vocabSizePadded);
    }
    else
    {
        size_t smemSize = sizeof(float) * batchSize;
        batchApplyTemperaturePenalty<T>
            <<<grid, block, smemSize, stream>>>(logits, bias, temperatures, batchSize, vocabSize, vocabSizePadded);
    }
}

template void invokeBatchApplyTemperaturePenalty(float* logits, const float* bias, const float* temperatures,
    const int batchSize, const int vocabSize, const int vocabSizePadded, hipStream_t stream);

template void invokeBatchApplyTemperaturePenalty(half* logits, const half* bias, const float* temperatures,
    const int batchSize, const int vocabSize, const int vocabSizePadded, hipStream_t stream);

template <typename T, RepetitionPenaltyType penaltyType>
__global__ void batchApplyRepetitionPenalty(T* logits, const float* penalties, const int** outputIds,
    const int* sequenceLengths, const int batchSize, const int vocabSize, const int maxSeqLen)
{
    extern __shared__ float penaltyLogits[];
    int* penaltyIndices = (int*) (penaltyLogits + maxSeqLen);
    const int batchIdx = blockIdx.x;
    const float penalty = penalties[batchIdx];
    const int currentStep = sequenceLengths[batchIdx];

    logits += batchIdx * vocabSize;

    // Phase 1. Find indices to penalize and keep the penalized values.
    // A vocab id can appear multiple times but should be penalized once.
    for (int index = threadIdx.x; index < currentStep; index += blockDim.x)
    {
        // outputIds shape: (batchSize, input_len + output_len)
        int penaltyIndex = outputIds[batchIdx][blockIdx.y * maxSeqLen + index];
        assert(penaltyIndex < vocabSize);
        penaltyIndices[index] = penaltyIndex;
        float logit = (float) logits[penaltyIndex];
        if (penaltyType == RepetitionPenaltyType::Additive)
        {
            penaltyLogits[index] = logit - penalty;
        }
        else if (penaltyType == RepetitionPenaltyType::Multiplicative)
        {
            penaltyLogits[index] = logit < 0.0f ? logit * penalty : logit / penalty;
        }
        else if (penaltyType == RepetitionPenaltyType::None)
        {
            penaltyLogits[index] = logit;
        }
        else
        {
            // Unsupported type
            assert(false);
        }
    }

    if (blockDim.x > 32)
    {
        __syncthreads();
    }

    // Phase 2. Replace a logit value by the penalized one.
    for (int index = threadIdx.x; index < currentStep; index += blockDim.x)
    {
        logits[penaltyIndices[index]] = penaltyLogits[index];
    }
}

template <typename T>
void invokeBatchApplyRepetitionPenalty(T* logits, const float* penalties, const int** outputIds,
    const int* sequenceLengths, const int batchSize, const int vocabSize, RepetitionPenaltyType penaltyType,
    int maxSeqLen, hipStream_t stream)
{
    TLLM_LOG_DEBUG("%s start", __PRETTY_FUNCTION__);
    dim3 block(min(maxSeqLen, 1024));
    dim3 grid(batchSize);
    // FIXME(nkorobov): with long sequences we might hit upper smem limit
    size_t smemSize = maxSeqLen * (sizeof(float) + sizeof(int));
    if (penaltyType == RepetitionPenaltyType::Additive)
    {
        if (smemSize >= 46 * 1024)
        {
            /* Set 46KB threshold here because we have to take static/driver shared memory into consideration. */
            hipError_t res = hipFuncSetAttribute(reinterpret_cast<const void*>(batchApplyRepetitionPenalty<T), RepetitionPenaltyType::Additive>,
                hipFuncAttributeMaxDynamicSharedMemorySize, smemSize);
            TLLM_CHECK_WITH_INFO(res == hipSuccess,
                "Sequence Length is too long for the batchApplyRepetitionPenalty kernel (not enough shared memory).");
        }
        batchApplyRepetitionPenalty<T, RepetitionPenaltyType::Additive><<<grid, block, smemSize, stream>>>(
            logits, penalties, outputIds, sequenceLengths, batchSize, vocabSize, maxSeqLen);
    }
    else if (penaltyType == RepetitionPenaltyType::Multiplicative)
    {
        if (smemSize >= 46 * 1024)
        {
            /* Set 46KB threshold here because we have to take static/driver shared memory into consideration. */
            hipError_t res
                = hipFuncSetAttribute(reinterpret_cast<const void*>(batchApplyRepetitionPenalty<T), RepetitionPenaltyType::Multiplicative>,
                    hipFuncAttributeMaxDynamicSharedMemorySize, smemSize);
            TLLM_CHECK_WITH_INFO(res == hipSuccess,
                "Sequence Length is too long for the batchApplyRepetitionPenalty kernel (not enough shared memory).");
        }
        batchApplyRepetitionPenalty<T, RepetitionPenaltyType::Multiplicative><<<grid, block, smemSize, stream>>>(
            logits, penalties, outputIds, sequenceLengths, batchSize, vocabSize, maxSeqLen);
    }
    else if (penaltyType == RepetitionPenaltyType::None)
    {
        // do nothing
    }
}

template void invokeBatchApplyRepetitionPenalty(float* logits, const float* penalties, const int** outputIds,
    const int* sequenceLengths, const int batchSize, const int vocabSize, RepetitionPenaltyType penaltyType,
    int maxSeqLen, hipStream_t stream);

template void invokeBatchApplyRepetitionPenalty(half* logits, const float* penalties, const int** outputIds,
    const int* sequenceLengths, const int batchSize, const int vocabSize, RepetitionPenaltyType penaltyType,
    int maxSeqLen, hipStream_t stream);

template <typename T>
__global__ void batchApplyMinLengthPenalty(T* logits, const int* minLengths, const int* endIds,
    const int* sequenceLengths, const int* contextLengths, const int vocabSizePaddeded)
{
    int bid = threadIdx.x + blockIdx.x * blockDim.x; // batch index
    auto const contextLength{contextLengths == nullptr ? 0 : contextLengths[bid]};
    // This kernel is called before sequenceLengths is incrememnted.
    // We need +1 because sequenceLengths = contextLength + numGenTokens - 1, which is equal to the length of k/v
    // caches.
    if (sequenceLengths[bid] + 1 - contextLength < minLengths[bid])
    {
        T maskVal = (std::is_same<T, half>::value) ? -65504.0f : -FLT_MAX;
        logits[bid * vocabSizePaddeded + endIds[bid]] = maskVal;
    }
}

template <typename T>
void invokeMinLengthPenalty(T* logits, const int* minLengths, const int* endIds, const int* sequneceLengths,
    const int* contextLengths, const int batchSize, const int vocabSizePaddeded, hipStream_t stream)

{
    const int blockSize = min(batchSize, 1024);
    const int gridSize = (batchSize + blockSize - 1) / blockSize;
    batchApplyMinLengthPenalty<<<gridSize, blockSize, 0, stream>>>(
        logits, minLengths, endIds, sequneceLengths, contextLengths, vocabSizePaddeded);
}

template void invokeMinLengthPenalty(float* logits, const int* minLengths, const int* endIds,
    const int* sequneceLengths, const int* contextLengths, const int batchSize, const int vocabSizePaddeded,
    hipStream_t stream);

template void invokeMinLengthPenalty(half* logits, const int* minLengths, const int* endIds, const int* sequneceLengths,
    const int* contextLengths, const int batchSize, const int vocabSizePaddeded, hipStream_t stream);

} // namespace kernels
} // namespace tensorrt_llm
